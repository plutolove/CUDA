#include "matrix_perfmance.h"

float testPerformance(dim3 grid, dim3 block, GemmType gemm, float* a, float* b,
                      float* c, int M, int N, int K, int repeat) {
  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start);

  for (int i = 0; i < repeat; i++) {
    gemm<<<grid, block>>>(a, b, c, M, N, K);
  }

  hipEventRecord(end);
  hipEventSynchronize(end);

  float msec, sec;
  hipEventElapsedTime(&msec, start, end);
  sec = msec / 1000.0 / repeat;

  return sec;
}
