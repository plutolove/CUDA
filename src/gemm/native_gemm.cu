#include "hip/hip_runtime.h"
#include <cstdio>

#include "common/macro.h"

__global__ void naiveSgemm(float* __restrict__ a, float* __restrict__ b,
                           float* __restrict__ c, const int M, const int N,
                           const int K) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  // a -> M * K
  // b -> K * N
  // c -> M * N
  if (x < M && y < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += a[x * K + k] * b[k * N + y];
    }
    c[x * N + y] = sum;
  }
}

__global__ void SgemmCoalescing(float* __restrict__ a, float* __restrict__ b,
                                float* __restrict__ c, const int M, const int N,
                                const int K) {
  auto thread_idx = blockDim.x * threadIdx.y + threadIdx.x;
  int x = blockIdx.x * blockDim.x + thread_idx / blockDim.x;
  int y = blockIdx.y * blockDim.y + thread_idx % blockDim.x;
  // a -> M * K
  // b -> K * N
  // c -> M * N
  if (x < M && y < N) {
    float sum = 0.0;
    for (int k = 0; k < K; k++) {
      sum += a[x * K + k] * b[k * N + y];
    }
    c[x * N + y] = sum;
  }
}
