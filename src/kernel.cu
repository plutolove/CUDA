
#include <hip/hip_runtime.h>
#include <iostream>

#define OFFSET(row, col, ld) ((row) * (ld) + (col))

const int BM = 32, BN = 32;

__global__ void naiveSgemm(float* __restrict__ a, float* __restrict__ b,
                           float* __restrict__ c, const int M, const int N,
                           const int K) {
    int n = blockIdx.x * blockDim.x + threadIdx.x;
    int m = blockIdx.y * blockDim.y + threadIdx.y;
    if (m < M && n < N) {
        float psum = 0.0;
        // #pragma unroll
        for (int k = 0; k < K; k++) {
            psum += a[OFFSET(m, k, K)] * b[OFFSET(k, n, N)];
        }
        c[OFFSET(m, n, N)] = psum;
    }
}

void naiveSgemm_cpp(float* a, float* b, float* c, int M, int N, int K) {
    dim3 blockDim(BN, BM);
    dim3 gridDim((N + BN - 1) / BN, (M + BM - 1) / BM);
    naiveSgemm<<<gridDim, blockDim>>>(a, b, c, M, N, K);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        std::cerr << "CUDA kernel launch error: " << hipGetErrorString(err)
                  << std::endl;
    }
    hipDeviceSynchronize();
}

float testPerformance(void (*gemm)(float*, float*, float*, int, int, int),
                      float* d_a, float* d_b, float* d_c, int M, int N, int K,
                      int repeat) {
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    hipEventRecord(start);

    for (int i = 0; i < repeat; i++) {
        gemm(d_a, d_b, d_c, M, N, K);
    }

    hipEventRecord(end);
    hipEventSynchronize(end);

    float msec, sec;
    hipEventElapsedTime(&msec, start, end);
    sec = msec / 1000.0 / repeat;

    return sec;
}
